#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <tuple>

#include "../common/utils_flat_float.h"
#include "../common/display_utils_flat_float.h"
#include "../common/config_reader.h"
#include "../common/time_utils.h"

#define GE(ans)                               \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

__global__ void thermodynamics_cuda(float *matrix, float *next_matrix, int row_size)
{

    // float diff;
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int x = index / row_size;
    int y = index % row_size;
    if ((x == 0) || (x == row_size - 1) || (y == 0) || (y == row_size - 1))
    {
        next_matrix[index] = matrix[index];
    }
    else
    {

        next_matrix[index] = (matrix[index - 1] +
                              matrix[index + 1] +
                              matrix[index - row_size] +
                              matrix[index + row_size]) /
                             4;
    }
    // diff = next_matrix[loc] - matrix[loc];
}

int main(int argc, char **argv)
{

    hipDeviceSynchronize();
    double start_time = getTime();
    string config_location = "config.ini";
    if (argc == 2)
    {
        // argv[0] - program name
        config_location = argv[1];
    }

    Configuration config = read_config(config_location);
    printf("config read time: %.6f\n", (getTime() - start_time));

    auto MATRIX_DIMENTION = config.matrix.dimention;
    auto MAX_MATRIX_VALUE = (float)config.matrix.max_value;
    auto MAX_ITERATION_COUNT = config.calculation.max_iteration_count;
    auto DRAW_FREQUENCY = config.drawing.draw_frequency;
    auto USE_ABS_SCALE = config.drawing.use_abs_scale;

    float *matrix = generate_matrix(MATRIX_DIMENTION, MAX_MATRIX_VALUE);
    float *result_matrix = new float[MATRIX_DIMENTION * MATRIX_DIMENTION];

    if (DRAW_FREQUENCY > 0)
    {
        save_to_file(matrix, MATRIX_DIMENTION, MAX_MATRIX_VALUE, 0, USE_ABS_SCALE);
    }
    float *cuda_matrix;
    float *cuda_result_matrix;
    int size = MATRIX_DIMENTION * MATRIX_DIMENTION * sizeof(float);
    GE(hipMalloc((void **)&cuda_matrix, size));
    GE(hipMalloc((void **)&cuda_result_matrix, size));
    GE(hipMemcpy(cuda_matrix, matrix, size, hipMemcpyHostToDevice));
    GE(hipMemcpy(cuda_result_matrix, result_matrix, size, hipMemcpyHostToDevice));

    // dim3 threads(tx, ty);
    // dim3 blocks(bx, by);
    int SPLIT = 256;
    printf("using %d %d data split \n", (MATRIX_DIMENTION * MATRIX_DIMENTION + SPLIT - 1) / SPLIT, SPLIT);

    for (int i = 1; i < MAX_ITERATION_COUNT; i++)
    {
        thermodynamics_cuda<<<(MATRIX_DIMENTION * MATRIX_DIMENTION + SPLIT - 1) / SPLIT, SPLIT>>>(cuda_matrix, cuda_result_matrix, MATRIX_DIMENTION);
        swap(cuda_matrix, cuda_result_matrix);
        if (DRAW_FREQUENCY > 0 && i % DRAW_FREQUENCY == 0)
        {
            GE(hipMemcpy(matrix, cuda_result_matrix, size, hipMemcpyDeviceToHost));
            save_to_file(matrix, MATRIX_DIMENTION, MAX_MATRIX_VALUE, i, USE_ABS_SCALE);
        }
    }
    GE(hipMemcpy(matrix, cuda_result_matrix, size, hipMemcpyDeviceToHost));
    hipDeviceSynchronize();
    double end_time = getTime();
    printf("execution time: %.3f\n", end_time - start_time);
    hipFree(cuda_matrix);
    hipFree(cuda_result_matrix);
}
