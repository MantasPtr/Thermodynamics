#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <tuple>

#include "../common/utils_flat.h"
#include "../common/display_utils_flat.h"
#include "../common/config_reader.h"
#include "../common/time_utils.h"

#define GE(ans)                               \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

__global__ void thermodynamics_cuda(double *matrix, double *next_matrix, double *diff, int row_size, bool use_diff)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int x = index / row_size;
    int y = index % row_size;
    if ((x == 0) || (x == row_size - 1) || (y == 0) || (y == row_size - 1))
    {
        next_matrix[index] = matrix[index];
    }
    else
    {

        next_matrix[index] = (matrix[index - 1] +
                              matrix[index + 1] +
                              matrix[index - row_size] +
                              matrix[index + row_size]) /
                             4;
    }
    if (use_diff)
    {
        diff[index] = next_matrix[index] - matrix[index];
    }
}

double largestElement(double *matrix, int row_size)
{
    int size = row_size * row_size;
    double max = -1;
    for (int i = 1; i < size; i++)
        if (matrix[i] > max)
            max = matrix[i];
    return max;
}

int main(int argc, char **argv)
{

    hipDeviceSynchronize();
    double start_time = getTime();
    string config_location = "config.ini";
    if (argc == 2)
    {
        // argv[0] - program name
        config_location = argv[1];
    }

    Configuration config = read_config(config_location);
    printf("config read time: %.6f\n", getTime() - start_time);

    auto MATRIX_DIMENTION = config.matrix.dimention;
    auto MAX_MATRIX_VALUE = config.matrix.max_value;
    auto MAX_ITERATION_COUNT = config.calculation.max_iteration_count;
    auto DRAW_FREQUENCY = config.drawing.draw_frequency;
    auto USE_ABS_SCALE = config.drawing.use_abs_scale;
    auto MAX_DELTA = config.calculation.delta;
    auto CUDA_SPLIT = config.calculation.cuda_split;
    bool USE_DIFF = MAX_DELTA > 0.0;

    double *matrix = generate_matrix(MATRIX_DIMENTION, MAX_MATRIX_VALUE);
    double *result_matrix = new double[MATRIX_DIMENTION * MATRIX_DIMENTION];
    double *diff_matrix = new double[MATRIX_DIMENTION * MATRIX_DIMENTION];

    if (DRAW_FREQUENCY > 0)
    {
        save_to_file(matrix, MATRIX_DIMENTION, MAX_MATRIX_VALUE, 0, USE_ABS_SCALE);
    }
    double *cuda_matrix;
    double *cuda_result_matrix;
    double *cuda_diff_matrix;
    int size = MATRIX_DIMENTION * MATRIX_DIMENTION * sizeof(double);
    GE(hipMalloc((void **)&cuda_matrix, size));
    GE(hipMalloc((void **)&cuda_result_matrix, size));
    GE(hipMemcpy(cuda_matrix, matrix, size, hipMemcpyHostToDevice));
    GE(hipMemcpy(cuda_result_matrix, result_matrix, size, hipMemcpyHostToDevice));
    if (USE_DIFF)
    {
        GE(hipMalloc((void **)&cuda_diff_matrix, size));
        GE(hipMemcpy(cuda_diff_matrix, diff_matrix, size, hipMemcpyHostToDevice));
    }
    int SPLIT = CUDA_SPLIT;
    printf("using %d %d data split \n", (MATRIX_DIMENTION * MATRIX_DIMENTION + SPLIT - 1) / SPLIT, SPLIT);
    int i = 0;
    double delta = MAX_MATRIX_VALUE;
    while (i < MAX_ITERATION_COUNT && delta > MAX_DELTA)
    {
        i++;
        thermodynamics_cuda<<<(MATRIX_DIMENTION * MATRIX_DIMENTION + SPLIT - 1) / SPLIT, SPLIT>>>(cuda_matrix, cuda_result_matrix, cuda_diff_matrix, MATRIX_DIMENTION, USE_DIFF);
        swap(cuda_matrix, cuda_result_matrix);
        if (DRAW_FREQUENCY > 0 && i % DRAW_FREQUENCY == 0)
        {
            GE(hipMemcpy(matrix, cuda_result_matrix, size, hipMemcpyDeviceToHost));
            save_to_file(matrix, MATRIX_DIMENTION, MAX_MATRIX_VALUE, i, USE_ABS_SCALE);
        }
        if ((USE_DIFF) && (i % 10000 == 0))
        {
            GE(hipMemcpy(diff_matrix, cuda_diff_matrix, size, hipMemcpyDeviceToHost));
            delta = largestElement(diff_matrix, MATRIX_DIMENTION);
            printf("iteration %d delta: %.7f\n", i, delta);
        }
    }
    GE(hipMemcpy(matrix, cuda_result_matrix, size, hipMemcpyDeviceToHost));
    GE(hipDeviceSynchronize());
    double end_time = getTime();
    printf("iteration count: %d\n", i);
    printf("execution time: %.3f\n", end_time - start_time);
    hipFree(cuda_matrix);
    hipFree(cuda_result_matrix);
}
